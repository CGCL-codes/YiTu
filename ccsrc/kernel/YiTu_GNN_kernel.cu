#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

#define WARP_SIZE 32

__global__ void warmup() {}

__device__ inline void atomicAdd_F(float *address, float value)
{
    float old = value;
    while ((old = atomicExch(address, atomicExch(address, 0.0f) + old)) != 0.0f)
        ;
}

template <typename scalar_t>
__global__ void SAG_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock);

template <typename scalar_t>
__global__ void spmm_forward_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock);

template <typename scalar_t>
__global__ void spmm_forward_cuda_kernel_gin(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    float epsilon,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock);

template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock);

template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel_gin(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    float epsilon,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock);

////////////////////////////////////////////
//
// Basic Scatter-And-Gather kernel.
//
////////////////////////////////////////////
torch::Tensor SAG_cuda(
    torch::Tensor input,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    torch::Tensor degrees,
    torch::Tensor part_pointers,
    torch::Tensor part2Node,
    int partSize,
    int dimWorker,
    int warpPerBlock)
{
    auto output = torch::zeros_like(input);

    const int num_nodes = output.size(0);
    const int dim = output.size(1);
    const int num_parts = part2Node.size(0);

    const int block = warpPerBlock * WARP_SIZE;
    const int grid = (num_parts * WARP_SIZE + block - 1) / block;
    int shared_memory = partSize * warpPerBlock * sizeof(int) + warpPerBlock * dim * sizeof(float);

    // printf("grid: %d, block: %d, shared_memory: %d\n", grid, block, shared_memory);
    // printf("dim: %d, num_nodes: %d, num_parts: %d\n", dim, num_nodes, num_parts);
    // printf("dimWorker: %d\n", dimWorker);
    // #define PROFILE 200

#ifdef PROFILE
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < PROFILE; i++)
    {
        warmup<<<1, 1>>>();
    }
    hipEventRecord(start, 0);

    for (int i = 0; i < PROFILE; i++)
#endif
        AT_DISPATCH_FLOATING_TYPES(input.type(), "Scatter_and_Gather", ([&]
                                                                        { SAG_cuda_kernel<scalar_t><<<grid, block, shared_memory>>>(
                                                                              output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                              input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                              row_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                              column_index.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                              degrees.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
                                                                              part_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                              part2Node.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                              num_nodes,
                                                                              dim,
                                                                              num_parts,
                                                                              partSize,
                                                                              dimWorker,
                                                                              warpPerBlock); }));

#ifdef PROFILE
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gflop = 2 * column_index.size(0) / 1e6 * dim;
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("TC-GNN -- Time (ms): %.3f, GFLOPs: %.3f\n", milliseconds / PROFILE, gflop / (milliseconds / PROFILE));
    printf("\n================================\n");
#endif

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    return output;
}

template <typename scalar_t>
__global__ void SAG_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x; // global thread-id
    int warpId = tid / WARP_SIZE;                    // global warp-id
    int block_warpId = threadIdx.x / WARP_SIZE;      // block warp-id
    int laneid = threadIdx.x % WARP_SIZE;            // warp thread-id -- laneid

    extern __shared__ int part_meta[];                                     // part information.
    int *partial_ids = part_meta;                                          // caching ids
    float *partial_results = (float *)&part_meta[partSize * warpPerBlock]; // caching partial results.

    if (warpId < num_parts)
    {

        int srcId = part2Node[warpId];           // aggregated source node
        int partBeg = part_pointers[warpId];     // partitioning pointer start
        int partEnd = part_pointers[warpId + 1]; // part pointer end

        // Cache the part neighbors.
        const int pindex_base = block_warpId * partSize;
        // #pragma unroll
        for (int nidx = partBeg + laneid; nidx < partEnd; nidx += dimWorker)
        {
            // printf("1--pindex_base: %d, laneid: %d\n", pindex_base, laneid);
            partial_ids[pindex_base + nidx - partBeg] = column_index[nidx];
            // if(partial_ids[pindex_base + laneid]  >= num_nodes || partial_ids[pindex_base + laneid]  < 0) printf("---- partial_ids: %d\n", partial_ids[pindex_base + laneid] );
        }

        __syncwarp();

        // Neighbor aggregation within each part
        const int presult_base = block_warpId * dim;
        for (int nIdx = 0; nIdx < partEnd - partBeg; nIdx++)
        {
            // if (laneid == 0) printf("2--pindex_base: %d, nIdx: %d\n", pindex_base, nIdx);
            int nid = partial_ids[pindex_base + nIdx];
            // if(nid >= num_nodes || nid < 0) printf("Error nid: %d\n", nid);

            // Initialize shared memory for partial results
            if (nIdx == 0)
                if (laneid < dimWorker)
#pragma unroll
                    for (int d = laneid; d < dim; d += dimWorker)
                    {
                        partial_results[presult_base + d] = 0.0f;
                    }

            if (laneid < dimWorker)
#pragma unroll
                for (int d = laneid; d < dim; d += dimWorker)
                {
                    partial_results[presult_base + d] += input[nid][d];
                }
        }

        // output the result to global memory from the shared memory
        if (laneid < dimWorker)
#pragma unroll
            for (int d = laneid; d < dim; d += dimWorker)
            {
                atomicAdd_F((float *)&output[srcId][d], partial_results[presult_base + d]);
            }
    }
}

////////////////////////////////////////////
//
// Foward Pass (GCN)  node update --> neighbor aggregation
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    torch::Tensor degrees,
    torch::Tensor part_pointers,
    torch::Tensor part2Node,
    int partSize,
    int dimWorker,
    int warpPerBlock)
{
    auto tmp = torch::mm(input, weight);
    // auto output = torch::zeros_like(tmp);
    auto output = torch::zeros({input.size(0), weight.size(1)}, torch::kCUDA);
    const int dim = output.size(1);
    const int num_nodes = output.size(0);
    const int num_parts = part2Node.size(0);

    const int block = warpPerBlock * WARP_SIZE;
    const int grid = (num_parts * WARP_SIZE + block - 1) / block;
    int shared_memory = partSize * warpPerBlock * sizeof(int) + warpPerBlock * dim * sizeof(float);

    // printf("grid: %d, block: %d\n", grid, block);
    // printf("dim: %d, num_nodes: %d, num_parts: %d\n", dim, num_nodes, num_parts);
    // printf("input: (%d, %d)\n", tmp.size(0), tmp.size(1));
    // printf("dimWorker: %d\n", dimWorker);
    // printf("shared_memory: %d\n", tmp.size(0), tmp.size(1));

    AT_DISPATCH_FLOATING_TYPES(input.type(), "spmm_cuda_forward", ([&]
                                                                   { spmm_forward_cuda_kernel<scalar_t><<<grid, block, shared_memory>>>(
                                                                         output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                         tmp.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                         row_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                         column_index.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                         degrees.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
                                                                         part_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                         part2Node.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                         num_nodes,
                                                                         dim,
                                                                         num_parts,
                                                                         partSize,
                                                                         dimWorker,
                                                                         warpPerBlock); }));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    return {output};
}

template <typename scalar_t>
__global__ void spmm_forward_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x; // global thread-id
    int warpId = tid / WARP_SIZE;                    // global warp-id
    int block_warpId = threadIdx.x / WARP_SIZE;      // block warp-id
    int laneid = threadIdx.x % WARP_SIZE;            // warp thread-id -- laneid

    extern __shared__ int part_meta[];                                     // part information.
    int *partial_ids = part_meta;                                          // caching ids
    float *partial_results = (float *)&part_meta[partSize * warpPerBlock]; // caching partial results.

    if (warpId < num_parts)
    {

        int srcId = part2Node[warpId];           // aggregated source node
        int partBeg = part_pointers[warpId];     // partitioning pointer start
        int partEnd = part_pointers[warpId + 1]; // part pointer end
        float src_norm = degrees[srcId];         // norm of the source node

        // Cache the part neighbors by all threads from a warp.
        const int pindex_base = block_warpId * partSize;
#pragma unroll
        for (int nidx = partBeg + laneid; nidx < partEnd; nidx += WARP_SIZE)
        {
            // if(column_index[nidx] >= num_nodes || column_index[nidx] < 0) printf("column_index: %d\n", column_index[nidx]);
            partial_ids[pindex_base + nidx - partBeg] = column_index[nidx];
        }

        // #pragma unroll
        // for (int nidx = partBeg; nidx < partEnd; nidx++){
        // //     if(column_index[nidx] >= num_nodes || column_index[nidx] < 0) printf("column_index: %d\n", column_index[nidx]);
        //     partial_ids[nidx - partBeg] = column_index[nidx];
        // }

        __syncwarp();

        // if (laneid == 0)
        // for (int nIdx = laneid; nIdx < partEnd - partBeg; nIdx++){
        // int nid = partial_ids[pindex_base + nIdx];
        // int nid = partial_ids[nIdx];
        // printf("verify nid - 111111: %d\n", nid);
        // if(nid >= num_nodes || nid < 0) printf("verify nid: %d\n", nid);
        // }

        // Neighbor aggregation within each part
        const int presult_base = block_warpId * dim;
        for (int nIdx = 0; nIdx < partEnd - partBeg; nIdx++)
        {
            int nid = partial_ids[pindex_base + nIdx];
            // int nid = partial_ids[nIdx];
            // if (laneid == 0)
            //     printf("verify nid - 222222: %d\n", nid);
            float degree_norm_inv = __fmaf_rn(src_norm, degrees[nid], 0);

            // Initialize shared memory for partial results
            if (nIdx == 0)
                if (laneid < dimWorker)
#pragma unroll
                    for (int d = laneid; d < dim; d += dimWorker)
                    {
                        partial_results[presult_base + d] = 0.0f;
                    }

            if (laneid < dimWorker)
#pragma unroll
                for (int d = laneid; d < dim; d += dimWorker)
                {
                    // if(nid >= num_nodes || nid < 0) printf("aggregation: %d\n", nid);
                    partial_results[presult_base + d] += __fmaf_rn(degree_norm_inv, input[nid][d], 0);
                    // partial_results[presult_base + d] += input[nid][d];
                }
        }

        // output the result to global memory from the shared memory
        if (laneid < dimWorker)
#pragma unroll
            for (int d = laneid; d < dim; d += dimWorker)
            {
                atomicAdd_F((float *)&output[srcId][d], partial_results[presult_base + d]);
            }
    }
}

////////////////////////////////////////////
//
// backward pass (GCN)
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_backward_cuda(
    torch::Tensor d_output,
    torch::Tensor X,
    torch::Tensor W,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    torch::Tensor degrees,
    torch::Tensor part_pointers,
    torch::Tensor part2Node,
    int partSize,
    int dimWorker,
    int warpPerBlock)
{

    auto d_input_prime = torch::zeros_like(d_output);

    const int dim = d_input_prime.size(1);
    const int num_nodes = d_input_prime.size(0);
    const int num_parts = part2Node.size(0);

    const int block = warpPerBlock * WARP_SIZE;
    const int grid = (num_parts * WARP_SIZE + block - 1) / block;
    // const int shared_memory = warpPerBlock * partSize * sizeof(int) + warpPerBlock * dim * sizeof(float);
    int shared_memory = partSize * warpPerBlock * sizeof(int) + warpPerBlock * dim * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES(d_output.type(), "spmm_cuda_backward", ([&]
                                                                       { spmm_backward_cuda_kernel<scalar_t><<<grid, block, shared_memory>>>(
                                                                             d_input_prime.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             row_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             column_index.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             degrees.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
                                                                             part_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             part2Node.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             num_nodes,
                                                                             dim,
                                                                             num_parts,
                                                                             partSize,
                                                                             dimWorker,
                                                                             warpPerBlock); }));

    // check for error
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    auto d_input = torch::mm(d_input_prime, W.transpose(0, 1));
    auto d_weight = torch::mm(X.transpose(0, 1), d_input_prime);

    return {d_input, d_weight};
}

template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = tid / WARP_SIZE;
    int block_warpId = threadIdx.x / WARP_SIZE;
    int laneid = threadIdx.x % WARP_SIZE;

    extern __shared__ int part_meta[];                                     // part information.
    int *partial_ids = part_meta;                                          // caching ids
    float *partial_results = (float *)&part_meta[partSize * warpPerBlock]; // caching partial results.

    if (warpId < num_parts)
    {

        const int srcId = part2Node[warpId];
        const int partBeg = part_pointers[warpId];
        const int partEnd = part_pointers[warpId + 1];
        float src_norm = degrees[srcId];

        const int pindex_base = block_warpId * partSize;
#pragma unroll
        for (int nid = partBeg + laneid; nid < partEnd; nid += WARP_SIZE)
        {
            partial_ids[pindex_base + nid - partBeg] = column_index[nid];
        }

        // #pragma unroll
        // for (int nidx = partBeg; nidx < partEnd; nidx++){
        // //     if(column_index[nidx] >= num_nodes || column_index[nidx] < 0) printf("column_index: %d\n", column_index[nidx]);
        //     partial_ids[nidx - partBeg] = column_index[nidx];
        // }

        __syncwarp();

        const int presult_base = block_warpId * dim;
        for (int nIdx = 0; nIdx < partEnd - partBeg; nIdx++)
        {
            int nid = partial_ids[pindex_base + nIdx];
            // int nid = partial_ids[nIdx];
            float degree_norm = __fmaf_rn(src_norm, degrees[nid], 0);

            if (nIdx == 0)
                if (laneid < dimWorker)
#pragma unroll
                    for (int d = laneid; d < dim; d += dimWorker)
                    {
                        partial_results[presult_base + d] = 0;
                    }

            if (laneid < dimWorker)
#pragma unroll
                for (int d = laneid; d < dim; d += dimWorker)
                {
                    partial_results[presult_base + d] += __fmaf_rn(degree_norm, d_output[nid][d], 0);
                }
        }

        if (laneid < dimWorker)
#pragma unroll
            for (int d = laneid; d < dim; d += dimWorker)
            {
                atomicAdd_F((float *)&d_input[srcId][d], partial_results[presult_base + d]);
            }
    }
}

////////////////////////////////////////////
//
// Foward Pass (GIN)
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_forward_cuda_gin(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    float epsilon,
    torch::Tensor part_pointers,
    torch::Tensor part2Node,
    int partSize,
    int dimWorker,
    int warpPerBlock)
{
    auto tmp = torch::zeros_like(input);
    const int dim = tmp.size(1);
    const int num_nodes = tmp.size(0);
    const int num_parts = part2Node.size(0);

    const int block = warpPerBlock * WARP_SIZE;
    const int grid = (num_parts * WARP_SIZE + block - 1) / block;
    const int shared_memory = warpPerBlock * partSize * sizeof(int) + warpPerBlock * dim * sizeof(float);

    // printf("grid: %d, block: %d\n", grid, block);
    // printf("dim: %d, num_nodes: %d, num_parts: %d\n", dim, num_nodes, num_parts);
    // printf("input: (%d, %d)\n", tmp.size(0), tmp.size(1));
    // printf("dimWorker: %d\n", dimWorker);
    // printf("warpPerBlock: %d, shared_memory: %d\n", warpPerBlock, shared_memory);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "spmm_cuda_forward_gin", ([&]
                                                                       { spmm_forward_cuda_kernel_gin<scalar_t><<<grid, block, shared_memory>>>(
                                                                             tmp.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             row_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             column_index.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             epsilon,
                                                                             part_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             part2Node.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                             num_nodes,
                                                                             dim,
                                                                             num_parts,
                                                                             partSize,
                                                                             dimWorker,
                                                                             warpPerBlock); }));

    auto output = torch::mm(tmp, weight);

    // check for error
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    return {output, tmp};
}

template <typename scalar_t>
__global__ void spmm_forward_cuda_kernel_gin(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    float epsilon,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x; // global thread-id
    int warpId = tid / WARP_SIZE;                    // global warp-id
    int block_warpId = threadIdx.x / WARP_SIZE;      // block warp-id
    int laneid = threadIdx.x % WARP_SIZE;            // warp thread-id -- laneid

    extern __shared__ int part_meta[];                                     // part information.
    int *partial_ids = part_meta;                                          // caching ids
    float *partial_results = (float *)&part_meta[partSize * warpPerBlock]; // caching partial results.

    if (warpId < num_parts)
    {

        int srcId = part2Node[warpId];           // aggregated source node
        int partBeg = part_pointers[warpId];     // partitioning pointer start
        int partEnd = part_pointers[warpId + 1]; // part pointer end

        // Cache the part neighbors.
        const int pindex_base = block_warpId * partSize;
#pragma unroll
        for (int nidx = partBeg + laneid; nidx < partEnd; nidx += dimWorker)
        {
            partial_ids[pindex_base + nidx - partBeg] = column_index[nidx];
        }

        __syncwarp();

        // Neighbor aggregation within each part
        const int presult_base = block_warpId * dim;
        for (int nIdx = 0; nIdx < partEnd - partBeg; nIdx++)
        {
            int nid = partial_ids[pindex_base + nIdx];

            // Initialize shared memory for partial results
            if (nIdx == 0)
                if (laneid < dimWorker)
#pragma unroll
                    for (int d = laneid; d < dim; d += dimWorker)
                    {
                        partial_results[presult_base + d] = 0.0f;
                    }

            if (laneid < dimWorker)
#pragma unroll
                for (int d = laneid; d < dim; d += dimWorker)
                {
                    partial_results[presult_base + d] += input[nid][d];
                }
        }

        // output the result to global memory from the shared memory
        if (laneid < dimWorker)
#pragma unroll
            for (int d = laneid; d < dim; d += dimWorker)
            {
                atomicAdd_F((float *)&output[srcId][d], epsilon * partial_results[presult_base + d]);
            }
    }
}

////////////////////////////////////////////
//
// backward pass (GIN)
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_backward_cuda_gin(
    torch::Tensor d_output,
    torch::Tensor X,
    torch::Tensor W,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    float epsilon,
    torch::Tensor part_pointers,
    torch::Tensor part2Node,
    int partSize,
    int dimWorker,
    int warpPerBlock)
{

    auto d_weight = torch::mm(X.transpose(0, 1), d_output);
    auto d_input_prime = torch::mm(d_output, W.transpose(0, 1));
    auto d_input = torch::zeros_like(d_input_prime);

    const int dim = d_input.size(1);
    const int num_nodes = d_input.size(0);
    const int num_parts = part2Node.size(0);

    const int block = warpPerBlock * WARP_SIZE;
    const int grid = (num_parts * WARP_SIZE + block - 1) / block;
    int shared_memory = partSize * warpPerBlock * sizeof(int) + warpPerBlock * dim * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES(d_output.type(), "spmm_cuda_backward_gin", ([&]
                                                                           { spmm_backward_cuda_kernel_gin<scalar_t><<<grid, block, shared_memory>>>(
                                                                                 d_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                                 d_input_prime.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                                 row_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                                 column_index.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                                 epsilon,
                                                                                 part_pointers.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                                 part2Node.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
                                                                                 num_nodes,
                                                                                 dim,
                                                                                 num_parts,
                                                                                 partSize,
                                                                                 dimWorker,
                                                                                 warpPerBlock); }));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    return {d_input, d_weight};
}

template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel_gin(
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> column_index,
    float epsilon,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> part2Node,
    const int num_nodes,
    const int dim,
    const int num_parts,
    const int partSize,
    const int dimWorker,
    const int warpPerBlock)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = tid / WARP_SIZE;
    int block_warpId = threadIdx.x / WARP_SIZE;
    int laneid = threadIdx.x % WARP_SIZE;

    extern __shared__ int part_meta[];                                     // part information.
    int *partial_ids = part_meta;                                          // caching ids
    float *partial_results = (float *)&part_meta[partSize * warpPerBlock]; // caching partial results.

    if (warpId < num_parts)
    {

        int srcId = part2Node[warpId];
        int partBeg = part_pointers[warpId];
        int partEnd = part_pointers[warpId + 1];

        const int pindex_base = block_warpId * partSize;
#pragma unroll
        for (int nid = partBeg + laneid; nid < partEnd; nid += dimWorker)
        {
            partial_ids[pindex_base + nid - partBeg] = column_index[nid];
        }

        __syncwarp();

        const int presult_base = block_warpId * dim;
        for (int nIdx = 0; nIdx < partEnd - partBeg; nIdx++)
        {
            int nid = partial_ids[pindex_base + nIdx];

            if (nIdx == 0)
#pragma unroll
                if (laneid < dimWorker)
                    for (int d = laneid; d < dim; d += dimWorker)
                    {
                        partial_results[presult_base + d] = 0;
                    }

            if (laneid < dimWorker)
#pragma unroll
                for (int d = laneid; d < dim; d += dimWorker)
                {
                    partial_results[presult_base + d] += d_output[nid][d];
                }
        }

        if (laneid < dimWorker)
#pragma unroll
            for (int d = laneid; d < dim; d += dimWorker)
            {
                atomicAdd_F((float *)&d_input[srcId][d], epsilon * partial_results[presult_base + d]);
            }
    }
}