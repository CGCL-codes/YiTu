#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"
#include "../shared/test.cuh"
#include "../shared/test.cu"


int main(int argc, char** argv)
{
	/*
	Test<int> test;
	cout << test.sum(20, 30) << endl;
	*/
	
	hipFree(0);

	ArgumentParser arguments(argc, argv, true, false);
	// cpu
	if (arguments.deviceID == -1)
	{
		std::string cmd = "./apps/BellmanFord " + arguments.input;
		FILE *pp = popen(cmd.data(), "r"); // build pipe
		if (!pp)
			return 1;
		// collect cmd execute result
		char tmp[1024];
		while (fgets(tmp, sizeof(tmp) * 1024, pp) != NULL)
			std::cout << tmp << std::endl; // can join each line as string
		pclose(pp);
		return 1;
	}
	Timer timer;
	timer.Start();
	
	GraphStructure graph;
	graph.ReadGraph(arguments.input);
	
	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
	
	//for(unsigned int i=0; i<100; i++)
	//	cout << graph.edgeList[i].end << " " << graph.edgeList[i].w8;
	GraphStates<uint> states(graph.num_nodes, true, false, false, graph.num_edges);
	states.ReadEdgeWeight(arguments.input + "w", graph.num_edges);

	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		states.value[i] = DIST_INFINITY;
		states.label1[i] = true;
		states.label2[i] = false;
	}
	states.value[arguments.sourceNode] = 0;
	//graph.label[arguments.sourceNode] = true;


	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_value, states.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label1, states.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label2, states.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	
	Subgraph subgraph(graph.num_nodes, graph.num_edges, true);
	
	SubgraphGenerator<uint> subgen(graph);
	
	subgen.generate(graph, states, subgraph);
	
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		states.label1[i] = false;
	}
	states.label1[arguments.sourceNode] = true;
	gpuErrorcheck(hipMemcpy(states.d_label1, states.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	

	Partitioner partitioner;
	
	timer.Start();
	
	uint gItr = 0;
	
	bool finished;
	bool *d_finished;
	bool all_finished;
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
		
	do
	{
		all_finished = true;
		gItr++;
		
		partitioner.partition(subgraph, subgraph.numActiveNodes);
		// a super iteration
		for(int i=0; i<partitioner.numPartitions; i++)
		{
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			gpuErrorcheck(hipMemcpy(subgraph.d_activeWeightList, subgraph.activeWeightList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(uint), hipMemcpyHostToDevice));
			hipDeviceSynchronize();

			//moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/512 + 1 , 512>>>(subgraph.d_activeNodes, states.d_label1, states.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			
			uint itr = 0;
			do
			{
				itr++;
				finished = true;
				gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				sssp_async<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
													partitioner.fromNode[i],
													partitioner.fromEdge[i],
													subgraph.d_activeNodes,
													subgraph.d_activeNodesPointer,
													subgraph.d_activeEdgeList,
													subgraph.d_activeWeightList,
													graph.d_outDegree,
													states.d_value,
													d_finished,
													(itr%2==1) ? states.d_label1 : states.d_label2,
													(itr%2==1) ? states.d_label2 : states.d_label1);

				hipDeviceSynchronize();
				gpuErrorcheck( hipPeekAtLastError() );	
				
				gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
				if(!finished) all_finished = false;			
			}while(!(finished));
			
			//cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;			
		}
		
		//subgen.generate(graph, states, subgraph);
			
	}while(!(all_finished));
	
	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime/1000 << " (s).\n";
	
	gpuErrorcheck(hipMemcpy(states.value, states.d_value, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost));
	
	utilities::PrintResults(states.value, min(30, graph.num_nodes));
		
	//for(int i=0; i<20; i++)
	//	cout << graph.value[i] << endl;
			
	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, states.value, graph.num_nodes);
}

