#include "hip/hip_runtime.h"
#include <cstdlib>

#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"


__global__ void bc_w(unsigned int numNodes,
							unsigned int from,
							unsigned int numPartitionedEdges,
							unsigned int *activeNodes,
							unsigned int *activeNodesPointer,
							OutEdge *edgeList,
							unsigned int *outDegree,
							unsigned int *dist,
							unsigned int *sigma,
							float *bc,
							bool *finished,
							int level)
{
	unsigned int tId = blockDim.x * blockIdx.x + threadIdx.x;

	if(tId < numNodes)
	{
		unsigned int id = activeNodes[from + tId];
		
		if(dist[id] != level)
			return;
			
		unsigned int sourceWeight = dist[id];

		unsigned int thisFrom = activeNodesPointer[from+tId]-numPartitionedEdges;
		unsigned int degree = outDegree[id];
		unsigned int thisTo = thisFrom + degree;
		
		//printf("******* %i\n", thisFrom);
		
		unsigned int finalDist;
		
		for(unsigned int i=thisFrom; i<thisTo; i++)
		{	
			//finalDist = sourceWeight + edgeList[i].w8;
			
			finalDist = sourceWeight + 1;
			if(finalDist < dist[edgeList[i].end])
			{
				atomicMin(&dist[edgeList[i].end] , level + 1);

				*finished = false;
			}
			if(dist[edgeList[i].end] == finalDist ) {
				atomicAdd(&sigma[edgeList[i].end] , sigma[id]);
			}
		}
	}
}


int main(int argc, char** argv)
{
	
	hipFree(0);

	ArgumentParser arguments(argc, argv, true, false);
	
	Timer timer;
	timer.Start();
	
	GraphStructure graph;
	graph.ReadGraph(arguments.input);
	
	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
	
	GraphStates<uint> states(graph.num_nodes, true, true, true);
	
	for (unsigned int i = 0; i < graph.num_nodes; i++)
	{
		states.value[i] = DIST_INFINITY;
		states.sigma[i] = 0;
		states.delta[i] = 0;
		states.label1[i] = true;
		states.label2[i] = false;
	}
	states.value[arguments.sourceNode] = 0;
	states.sigma[arguments.sourceNode] = 1;
	states.delta[arguments.sourceNode] = 0;


	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_value, states.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_sigma, states.sigma, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_delta, states.delta, graph.num_nodes * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label1, states.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label2, states.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	
	Subgraph subgraph(graph.num_nodes, graph.num_edges);
	SubgraphGenerator<uint> subgen(graph);
	
	subgen.generate(graph, states, subgraph);	
	for(unsigned int i=0; i<graph.num_nodes; i++)//仅将源顶点标记为活跃顶点
	{
		states.label1[i] = false;
	}
	states.label1[arguments.sourceNode] = true;
	gpuErrorcheck(hipMemcpy(states.d_label1, states.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));

	Partitioner partitioner;
	
	timer.Start();
	
	uint gItr = 0;
	unsigned int level = 0;
	unsigned int* d_level;
	bool finished;
	bool *d_finished;
	bool all_finished;
	gpuErrorcheck(hipMalloc(&d_level, sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	partitioner.partition(subgraph, subgraph.numActiveNodes);

	do
	{
		gItr++;
		all_finished = true;
		uint itr = 0;//分区迭代数
		//partitioner.partition(subgraph, subgraph.numActiveNodes);
		//cout << "partition number: " << partitioner.numPartitions << endl;
		// a super iteration
		for (int i = 0; i < partitioner.numPartitions; i++)
		{
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			//hipDeviceSynchronize();

			//moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels << <partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (subgraph.d_activeNodes, states.d_label1, states.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);

			//uint itr = 0;
			do
			{
				itr++;
				finished = true;
				gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
				bfs_async << < partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (partitioner.partitionNodeSize[i],
					partitioner.fromNode[i],
					partitioner.fromEdge[i],
					subgraph.d_activeNodes,
					subgraph.d_activeNodesPointer,
					subgraph.d_activeEdgeList,
					graph.d_outDegree,
					states.d_value,
					d_finished,
					(itr % 2 == 1) ? states.d_label1 : states.d_label2,
					(itr % 2 == 1) ? states.d_label2 : states.d_label1);
				hipDeviceSynchronize();
				gpuErrorcheck(hipPeekAtLastError());

				gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
				if (!finished) all_finished = false;
			} while (!(finished));
			//cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;
		}
		//subgen.generate(graph, states, subgraph);
	} while (!(all_finished));

	hipDeviceSynchronize();
	gpuErrorcheck(hipMemcpy(d_level, &level, sizeof(unsigned int), hipMemcpyHostToDevice));
	find_max << < graph.num_nodes / 512 + 1, 512 >> > (graph.num_nodes,
		states.d_value,
		d_level);
	hipDeviceSynchronize();
	gpuErrorcheck(hipPeekAtLastError());
	gpuErrorcheck(hipMemcpy(&level, d_level, sizeof(bool), hipMemcpyDeviceToHost));
	level++;
	cout << level << endl;

	for (unsigned int i = 0; i < graph.num_nodes; i++)
	{
		states.label1[i] = true;
		states.label2[i] = false;
	}
	gpuErrorcheck(hipMemcpy(states.d_label1, states.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label2, states.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	//subgen.generate(graph, states, subgraph);

	int start = 0;
	while (start < level) {
		//partitioner.partition(subgraph, subgraph.numActiveNodes);
		//cout<< level << "  " <<subgraph1.numActiveNodes<<" .   "<<partitioner1.numPartitions <<endl;
		for (int i = 0; i < partitioner.numPartitions; i++) {
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			//hipDeviceSynchronize();
			//moveUpLabels<<<partitioner1.partitionNodeSize[i]/512 + 1 , 512>>>(subgraph1.d_activeNodes, states.d_label1, states.d_label2, partitioner1.partitionNodeSize[i], partitioner1.fromNode[i]);

			bc_sigma_async << < partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (partitioner.partitionNodeSize[i],
				partitioner.fromNode[i],
				partitioner.fromEdge[i],
				subgraph.d_activeNodes,
				subgraph.d_activeNodesPointer,
				subgraph.d_activeEdgeList,
				graph.d_outDegree,
				states.d_value,
				states.d_sigma,
				start);

			gpuErrorcheck(hipDeviceSynchronize());
			gpuErrorcheck(hipPeekAtLastError());

		}
		start++;
	}

	//subgen.generate(graph, states, subgraph);

	while (level >= 1) {
		level--;

		//partitioner.partition(subgraph, subgraph.numActiveNodes);
		//cout<< level << "  " <<subgraph1.numActiveNodes<<" .   "<<partitioner1.numPartitions <<endl;
		for (int i = 0; i < partitioner.numPartitions; i++) {
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			//hipDeviceSynchronize();
			//moveUpLabels<<<partitioner1.partitionNodeSize[i]/512 + 1 , 512>>>(subgraph1.d_activeNodes, states.d_label1, states.d_label2, partitioner1.partitionNodeSize[i], partitioner1.fromNode[i]);

			bc_ndp << < partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (partitioner.partitionNodeSize[i],
				partitioner.fromNode[i],
				partitioner.fromEdge[i],
				subgraph.d_activeNodes,
				subgraph.d_activeNodesPointer,
				subgraph.d_activeEdgeList,
				graph.d_outDegree,
				states.d_value,
				states.d_sigma,
				states.d_delta,
				states.d_label1,
				states.d_label2,
				level);

			gpuErrorcheck(hipDeviceSynchronize());
			gpuErrorcheck(hipPeekAtLastError());

		}
		//subgen.generate(graph, states, subgraph);
	}

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime / 1000 << " (s).\n";

	gpuErrorcheck(hipMemcpy(states.value, states.d_value, graph.num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
	gpuErrorcheck(hipMemcpy(states.sigma, states.d_sigma, graph.num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
	gpuErrorcheck(hipMemcpy(states.delta, states.d_delta, graph.num_nodes * sizeof(float), hipMemcpyDeviceToHost));
	utilities::PrintResults(states.delta, min(100, graph.num_nodes));


	if (arguments.hasOutput)
		utilities::SaveResults(arguments.output, states.delta, graph.num_nodes);
}

