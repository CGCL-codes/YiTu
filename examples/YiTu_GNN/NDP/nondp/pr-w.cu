#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"
#include "../shared/test.cuh"
#include "../shared/test.cu"


int main(int argc, char** argv)
{
	
	hipFree(0);

	ArgumentParser arguments(argc, argv, true, false);
	// cpu
	if (arguments.deviceID == -1)
	{
		std::string cmd = "./apps/PageRank " + arguments.input;
		FILE *pp = popen(cmd.data(), "r"); // build pipe
		if (!pp)
			return 1;
		// collect cmd execute result
		char tmp[1024];
		while (fgets(tmp, sizeof(tmp) * 1024, pp) != NULL)
			std::cout << tmp << std::endl; // can join each line as string
		pclose(pp);
		return 1;
	}
	Timer timer;
	timer.Start();
	
	GraphStructure graph;
	graph.ReadGraph(arguments.input);
	
	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
	
	//for(unsigned int i=0; i<100; i++)
	//	cout << graph.edgeList[i].end << " " << graph.edgeList[i].w8;
	GraphStates<float> states(graph.num_nodes, false, true, false);
	
	float initPR = 0.15;
	float acc = 0.01;
	
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		states.delta[i] = initPR;
		states.value[i] = 0;
	}
	//graph.value[arguments.sourceNode] = 0;
	//graph.label[arguments.sourceNode] = true;


	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_value, states.value, graph.num_nodes * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_delta, states.delta, graph.num_nodes * sizeof(float), hipMemcpyHostToDevice));
	
	Subgraph subgraph(graph.num_nodes, graph.num_edges);
	
	SubgraphGenerator<float> subgen(graph);
	
	subgen.generate(graph, states, subgraph, acc);	

	Partitioner partitioner;
	
	timer.Start();
	
	uint gItr = 0;
	
	bool finished;
	bool *d_finished;
	bool all_finished;
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
		
	do
	{
		all_finished = true;
		gItr++;
		
		partitioner.partition(subgraph, subgraph.numActiveNodes);
		// a super iteration
		for(int i=0; i<partitioner.numPartitions; i++)
		{
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			hipDeviceSynchronize();

			//moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			//mixLabels<<<partitioner.partitionNodeSize[i]/512 + 1 , 512>>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			
			uint itr = 0;
			do
			{
				itr++;
				finished = true;
				gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				pr_async<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
													partitioner.fromNode[i],
													partitioner.fromEdge[i],
													subgraph.d_activeNodes,
													subgraph.d_activeNodesPointer,
													subgraph.d_activeEdgeList,
													graph.d_outDegree,
													states.d_value,
													states.d_delta,
													d_finished,
													acc);		
																						

				hipDeviceSynchronize();
				gpuErrorcheck( hipPeekAtLastError() );	
				
				gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
				if(!finished) all_finished = false;
			}while(!(finished));
			
			//cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;			
		}
		
		//subgen.generate(graph, states, subgraph, acc);
	}	while(!(all_finished));
	
	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime/1000 << " (s).\n";
	
	gpuErrorcheck(hipMemcpy(states.value, states.d_value, graph.num_nodes*sizeof(float), hipMemcpyDeviceToHost));
	
	utilities::PrintResults(states.value, min(30, graph.num_nodes));
	
		
	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, states.value, graph.num_nodes);
}

