#include "hip/hip_runtime.h"
#include "subgraph_generator.cuh"
#include "graph.cuh"
#include "subgraph.cuh"
#include "gpu_error_check.cuh"

const unsigned int NUM_THREADS = 64;

const unsigned int THRESHOLD_THREAD = 50000;

__global__ void prePrefix(unsigned int *activeNodesLabeling, unsigned int *activeNodesDegree, 
							unsigned int *outDegree, bool *label1, bool *label2, unsigned int numNodes)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < numNodes){
		activeNodesLabeling[id] = label1[id] || label2[id]; // label1 is always zero in sync
		//activeNodesLabeling[id] = label[id];
		//activeNodesLabeling[id] = 1;
		activeNodesDegree[id] = 0;
		if(activeNodesLabeling[id] == 1)
			activeNodesDegree[id] = outDegree[id];	
	}	
}

__global__ void prePrefix(unsigned int *activeNodesLabeling, unsigned int *activeNodesDegree, 
							unsigned int *outDegree, float *delta, unsigned int numNodes, float acc)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < numNodes){
		if(delta[id] > acc)
		{
			activeNodesLabeling[id] = 1;
		}
		else
		{
			activeNodesLabeling[id] = 0;
		}
		activeNodesDegree[id] = 0;
		if(activeNodesLabeling[id] == 1)
			activeNodesDegree[id] = outDegree[id];	
	}
}

__global__ void makeQueue(unsigned int *activeNodes, unsigned int *activeNodesLabeling,
							unsigned int *prefixLabeling, unsigned int numNodes)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < numNodes && activeNodesLabeling[id] == 1){
		activeNodes[prefixLabeling[id]] = id;
	}
}

__global__ void makeActiveNodesPointer(unsigned int *activeNodesPointer, unsigned int *activeNodesLabeling, 
											unsigned int *prefixLabeling, unsigned int *prefixSumDegrees, 
											unsigned int numNodes)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < numNodes && activeNodesLabeling[id] == 1){
		activeNodesPointer[prefixLabeling[id]] = prefixSumDegrees[id];
	}
}

// pthread
void dynamic(unsigned int tId,
				unsigned int numThreads,	
				unsigned int numActiveNodes,
				unsigned int *activeNodes,
				unsigned int *outDegree, 
				unsigned int *activeNodesPointer,
				unsigned int *nodePointer, 
				OutEdge *activeEdgeList,
				OutEdge *edgeList,
				uint* activeWeightList = NULL,
				uint* weightList = NULL)
{

	unsigned int chunkSize = numActiveNodes / numThreads;
	if(numActiveNodes % numThreads != 0)
	{
		chunkSize++;
	}
	unsigned int left, right;
	left = tId * chunkSize;
	right = min(left+chunkSize, numActiveNodes);	
	
	unsigned int thisNode;
	unsigned int thisDegree;
	unsigned int fromHere;
	unsigned int fromThere;

	for(unsigned int i=left; i<right; i++)
	{
		thisNode = activeNodes[i];
		thisDegree = outDegree[thisNode];
		fromHere = activeNodesPointer[i];
		fromThere = nodePointer[thisNode];
		for(unsigned int j=0; j<thisDegree; j++)
		{
			activeEdgeList[fromHere+j] = edgeList[fromThere+j];
			if (weightList != NULL)
				activeWeightList[fromHere + j] = weightList[fromThere + j];
		}
	}
	
}

template <class valueType>
SubgraphGenerator<valueType>::SubgraphGenerator(GraphStructure& graph)
{
	gpuErrorcheck(hipHostMalloc(&activeNodesLabeling, graph.num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipHostMalloc(&activeNodesDegree, graph.num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipHostMalloc(&prefixLabeling, graph.num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipHostMalloc(&prefixSumDegrees, (graph.num_nodes+1) * sizeof(unsigned int)));
	
	gpuErrorcheck(hipMalloc(&d_activeNodesLabeling, graph.num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_activeNodesDegree, graph.num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_prefixLabeling, graph.num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_prefixSumDegrees , (graph.num_nodes+1) * sizeof(unsigned int)));
}

template <class valueType>
void SubgraphGenerator<valueType>::FreeSubgraphGenerator()
{
	gpuErrorcheck(hipFree(d_activeNodesLabeling));
	gpuErrorcheck(hipFree(d_activeNodesDegree));
	gpuErrorcheck(hipFree(d_prefixLabeling));
	gpuErrorcheck(hipFree(d_prefixSumDegrees));
	gpuErrorcheck(hipHostFree(activeNodesLabeling));
	gpuErrorcheck(hipHostFree(activeNodesDegree));
	gpuErrorcheck(hipHostFree(prefixLabeling));
	gpuErrorcheck(hipHostFree(prefixSumDegrees));
}

template <class valueType>
void SubgraphGenerator<valueType>::generate(GraphStructure& graph, GraphStates<valueType>& states, Subgraph& subgraph, float acc)
{
	//std::chrono::time_point<std::chrono::system_clock> startDynG, finishDynG;
	//startDynG = std::chrono::system_clock::now();
	
	if (acc == -1)
		prePrefix << <graph.num_nodes / 512 + 1, 512 >> > (d_activeNodesLabeling, d_activeNodesDegree, graph.d_outDegree, states.d_label1, states.d_label2, graph.num_nodes);
	else
		prePrefix << <graph.num_nodes / 512 + 1, 512 >> > (d_activeNodesLabeling, d_activeNodesDegree, graph.d_outDegree, states.d_delta, graph.num_nodes, acc);
		
	thrust::device_ptr<unsigned int> ptr_labeling(d_activeNodesLabeling);
	thrust::device_ptr<unsigned int> ptr_labeling_prefixsum(d_prefixLabeling);
	
	subgraph.numActiveNodes = thrust::reduce(ptr_labeling, ptr_labeling + graph.num_nodes);
	//cout << "Number of Active Nodes = " << subgraph.numActiveNodes << endl;
				
	thrust::exclusive_scan(ptr_labeling, ptr_labeling + graph.num_nodes, ptr_labeling_prefixsum);
	
	makeQueue<<<graph.num_nodes/512+1, 512>>>(subgraph.d_activeNodes, d_activeNodesLabeling, d_prefixLabeling, graph.num_nodes);
	
	gpuErrorcheck(hipMemcpy(subgraph.activeNodes, subgraph.d_activeNodes, subgraph.numActiveNodes*sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	thrust::device_ptr<unsigned int> ptr_degrees(d_activeNodesDegree);
	thrust::device_ptr<unsigned int> ptr_degrees_prefixsum(d_prefixSumDegrees);
	
	thrust::exclusive_scan(ptr_degrees, ptr_degrees + graph.num_nodes, ptr_degrees_prefixsum);
	
	makeActiveNodesPointer<<<graph.num_nodes/512+1, 512>>>(subgraph.d_activeNodesPointer, d_activeNodesLabeling, d_prefixLabeling, d_prefixSumDegrees, graph.num_nodes);
	gpuErrorcheck(hipMemcpy(subgraph.activeNodesPointer, subgraph.d_activeNodesPointer, subgraph.numActiveNodes*sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	unsigned int numActiveEdges = 0;
	if(subgraph.numActiveNodes>0)
		numActiveEdges = subgraph.activeNodesPointer[subgraph.numActiveNodes-1] + graph.outDegree[subgraph.activeNodes[subgraph.numActiveNodes-1]];	
	
	unsigned int last = numActiveEdges;
	gpuErrorcheck(hipMemcpy(subgraph.d_activeNodesPointer+subgraph.numActiveNodes, &last, sizeof(unsigned int), hipMemcpyHostToDevice));
	
	gpuErrorcheck(hipMemcpy(subgraph.activeNodesPointer, subgraph.d_activeNodesPointer, (subgraph.numActiveNodes+1)*sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	
	//finishDynG = std::chrono::system_clock::now();
	//std::chrono::duration<double> elapsed_seconds_dyng = finishDynG-startDynG;
	//std::time_t finish_time_dyng = std::chrono::system_clock::to_time_t(finishDynG);
	//std::cout << "Dynamic GPU Time = " << elapsed_seconds_dyng.count() << std::endl;
	
	//td::chrono::time_point<std::chrono::system_clock> startDynC, finishDynC;
	//startDynC = std::chrono::system_clock::now();
	
	unsigned int numThreads = NUM_THREADS;

	if(subgraph.numActiveNodes < THRESHOLD_THREAD)
		numThreads = 1;

	thread runThreads[numThreads];
	
	for(unsigned int t=0; t<numThreads; t++)
	{

		runThreads[t] = thread(dynamic,
								t,
								numThreads,
								subgraph.numActiveNodes,
								subgraph.activeNodes,
								graph.outDegree, 
								subgraph.activeNodesPointer,
								graph.nodePointer, 
								subgraph.activeEdgeList,
								graph.edgeList,
								subgraph.activeWeightList,
								states.edgeWeight);

	}
		
	for(unsigned int t=0; t<numThreads; t++)
		runThreads[t].join();
	
	//finishDynC = std::chrono::system_clock::now();
	//std::chrono::duration<double> elapsed_seconds_dync = finishDynC-startDynC;
	//std::time_t finish_time_dync = std::chrono::system_clock::to_time_t(finishDynC);
	//std::cout << "Dynamic CPU Time = " << elapsed_seconds_dync.count() << std::endl;
	
}

template class SubgraphGenerator<uint>;
template class SubgraphGenerator<float>;

