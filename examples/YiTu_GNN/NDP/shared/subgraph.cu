
#include "subgraph.cuh"
#include "gpu_error_check.cuh"
#include "graph.cuh"
#include <hip/hip_runtime_api.h>


Subgraph::Subgraph(uint num_nodes, uint num_edges, bool hasEdgeWeight)
{
	hipProfilerStart();
	hipError_t error;
	hipDeviceProp_t dev;
	int deviceID;
	hipGetDevice(&deviceID);
	error = hipGetDeviceProperties(&dev, deviceID);
	if(error != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipProfilerStop();
	
	this->hasWeight = hasEdgeWeight;
	if (hasEdgeWeight)
		max_partition_size = 0.9 * (dev.totalGlobalMem - 8 * 4 * num_nodes) / (sizeof(OutEdge) + sizeof(uint));
	else
		max_partition_size = 0.9 * (dev.totalGlobalMem - 8 * 4 * num_nodes) / sizeof(OutEdge);
	//max_partition_size = 1000000000;
	
	if(max_partition_size > DIST_INFINITY)
		max_partition_size = DIST_INFINITY;
	
	//cout << "Max Partition Size: " << max_partition_size << endl;
	
	this->num_nodes = num_nodes;
	this->num_edges = num_edges;
	
	gpuErrorcheck(hipHostMalloc(&activeNodes, num_nodes * sizeof(uint)));
	gpuErrorcheck(hipHostMalloc(&activeNodesPointer, (num_nodes+1) * sizeof(uint)));
	gpuErrorcheck(hipHostMalloc(&activeEdgeList, num_edges * sizeof(OutEdge)));
	
	gpuErrorcheck(hipMalloc(&d_activeNodes, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_activeNodesPointer, (num_nodes+1) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_activeEdgeList, (max_partition_size) * sizeof(OutEdge)));
	
	if (hasEdgeWeight)
	{
		gpuErrorcheck(hipHostMalloc(&activeWeightList, num_edges * sizeof(uint)));
		gpuErrorcheck(hipMalloc(&d_activeWeightList, (max_partition_size) * sizeof(uint)));
	}
}

void Subgraph::FreeSubgraph()
{
	gpuErrorcheck(hipFree(d_activeNodes));
	gpuErrorcheck(hipFree(d_activeNodesPointer));
	gpuErrorcheck(hipFree(d_activeEdgeList));
	gpuErrorcheck(hipHostFree(activeNodes));
	gpuErrorcheck(hipHostFree(activeNodesPointer));
	gpuErrorcheck(hipHostFree(activeEdgeList));
	if (hasWeight)
	{
		gpuErrorcheck(hipFree(d_activeWeightList));
		gpuErrorcheck(hipHostFree(activeWeightList));
	}
}
// For initialization with one active node
//unsigned int numActiveNodes = 1;
//subgraph.activeNodes[0] = SOURCE_NODE;
//for(unsigned int i=graph.nodePointer[SOURCE_NODE], j=0; i<graph.nodePointer[SOURCE_NODE] + graph.outDegree[SOURCE_NODE]; i++, j++)
//	subgraph.activeEdgeList[j] = graph.edgeList[i];
//subgraph.activeNodesPointer[0] = 0;
//subgraph.activeNodesPointer[1] = graph.outDegree[SOURCE_NODE];
//gpuErrorcheck(hipMemcpy(subgraph.d_activeNodes, subgraph.activeNodes, numActiveNodes * sizeof(unsigned int), hipMemcpyHostToDevice));
//gpuErrorcheck(hipMemcpy(subgraph.d_activeNodesPointer, subgraph.activeNodesPointer, (numActiveNodes+1) * sizeof(unsigned int), hipMemcpyHostToDevice));


