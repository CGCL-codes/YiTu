#include "argument_parsing.cuh"


    
ArgumentParser::ArgumentParser(int argc, char **argv, bool canHaveSource, bool canHaveItrs)
{
	this->argc = argc;
	this->argv = argv;
	this->canHaveSource = canHaveSource;
	this->canHaveItrs = canHaveItrs;
	
	this->sourceNode = 0;
	this->deviceID = 0;
	this->numberOfItrs = 1;
	
	hasInput = false;
	hasSourceNode = false;
	hasOutput = false;
	hasDeviceID = false;
	hasNumberOfItrs = false;
	isDistributed = false;
	isSynchronize = false;

	Parse();
}
	
bool ArgumentParser::Parse()
{
	try
	{
		if(argc == 1)
		{
			cout << GenerateHelpString();
			exit(0);
		}
		
		if(argc == 2) 
			if ((strcmp(argv[1], "--help") == 0) || 
				(strcmp(argv[1], "-help") == 0) || 
				(strcmp(argv[1], "--h") == 0) || 
				(strcmp(argv[1], "-h") == 0))
			{
				cout << GenerateHelpString();
				exit(0);
			}
		
		if(argc%2 == 0)
		{
			cout << "\nThere was an error parsing command line arguments\n";
			cout << GenerateHelpString();
			exit(0);
		}
		
			
		for(int i=1; i<argc-1; i=i+2)
		{
			//argv[i]
			
			if (strcmp(argv[i], "--input") == 0) {
				input = string(argv[i+1]);
				hasInput = true;
			}
			else if (strcmp(argv[i], "--output") == 0) {
				output = string(argv[i+1]);
				hasOutput = true;
			}
			else if (strcmp(argv[i], "--source") == 0 && canHaveSource) {
				sourceNode = atoi(argv[i+1]);
				hasSourceNode = true;
			}
			else if (strcmp(argv[i], "--device") == 0) {
				deviceID = atoi(argv[i+1]);
				hasDeviceID = true;
				hipSetDevice(deviceID);
			}
			else if (strcmp(argv[i], "--iteration") == 0 && canHaveItrs) {
				numberOfItrs = atoi(argv[i+1]);
				hasNumberOfItrs = true;
			}
			else if (strcmp(argv[i], "--sync") == 0) {
				if (atoi(argv[i + 1]) == 1) isSynchronize = true;
			}
			else if (strcmp(argv[i], "--dist") == 0) {
				if (atoi(argv[i + 1]) == 1) isDistributed = true;
			}
			else
			{
				cout << "\nThere was an error parsing command line argument <" << argv[i] << ">\n";
				cout << GenerateHelpString();
				exit(0);
			}
		}
		
		if(hasInput)
			return true;
		else
		{
			cout << "\nInput graph file argument is required.\n";
			cout << GenerateHelpString();
			exit(0);
		}
	}
	catch( const std::exception& strException ) {
		std::cerr << strException.what() << "\n";
		GenerateHelpString();
		exit(0);
	}
	catch(...) {
		std::cerr << "An exception has occurred.\n";
		GenerateHelpString();
		exit(0);
	}
}

string ArgumentParser::GenerateHelpString(){
	string str = "\nRequired arguments:";
	str += "\n    [--input]: Input graph file. E.g., --input FacebookGraph.txt";
	str += "\nOptional arguments";
	if(canHaveSource)
		str += "\n    [--source]:  Begins from the source (Default: 0). E.g., --source 10";
	str += "\n    [--output]: Output file for results. E.g., --output results.txt";
	str += "\n    [--device]: Select GPU device (default: 0). E.g., --device 1";
	if(canHaveItrs)
		str += "\n    [--iteration]: Number of iterations (default: 1). E.g., --iterations 10";
	str += "\n\n";
	return str;
}

